#include <algorithm>
#include <iostream>     // std::cout
#include <fstream>      // std::ifstream
#include <vector>

#include "matrix.h"
#include "read_csv.h"

cu2rec::CudaCSRMatrix* readSparseMatrix(std::vector<Rating> *ratings, int rows, int cols) {
    //int *indptr = new int[ratings->size()];
    std::vector<int> indptr_vec;
    int *indices = new int[ratings->size()];
    float *data = new float[ratings->size()];
    int lastUser = -1;
    for(int i = 0; i < ratings->size(); ++i) {
        Rating r = ratings->at(i);
        if(r.userID != lastUser) {
            indptr_vec.push_back(r.userID);
            lastUser = r.userID;
        }
        indices[i] = r.itemID;
        data[i] = r.rating;

    }
    indptr_vec.push_back(ratings->size());
    int *indptr = indptr_vec.data();
    const int *indptr_c = const_cast<const int*>(indptr);
    const int *indices_c = const_cast<const int*>(indices);
    const float *data_c = const_cast<const float*>(data);
    cu2rec::CudaCSRMatrix* matrix = new cu2rec::CudaCSRMatrix(rows, cols, (int)(ratings->size()), indptr_c, indices_c, data_c);
    
    // Delete host arrays
    delete[] indptr;
    delete[] indices;
    delete[] data;
    return matrix;
}

int main(int argc, char **argv){
    int rows, cols;
    std::vector<Rating> ratings = readCSV(argv[1], &rows, &cols);
    printCSV(&ratings);
    cu2rec::CudaCSRMatrix* matrix = readSparseMatrix(&ratings, rows, cols);
}