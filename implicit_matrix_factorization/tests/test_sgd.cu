#include <assert.h>
#include <vector>

#include "../matrix.h"
#include "../read_csv.h"
#include "../sgd.h"

using namespace cu2rec;
using namespace std;

string filename = "../../data/test_ratings.csv";

void test_sgd() {
    // Initalize the input matrix
    int rows, cols;
    vector<Rating> ratings = readCSV(filename, &rows, &cols);
    CudaCSRMatrix* matrix = createSparseMatrix(&ratings, rows, cols);

    // Hyperparams
    int n_factors = 1;
    float learning_rate = 1e-3;

    // Initialize P and Q
    float *P = new float[rows * n_factors];
    for(int i = 0; i < rows; ++i) {
        P[i] = 1;
    }
    float *Q = new float[cols * n_factors];
    for(int i = 0; i < cols; ++i) {
        Q[i] = 1;
    }

    // Copy P and Q to device memory
    float *P_device, *P_device_target;
    int P_size = rows * n_factors * sizeof(float);
    hipMalloc(&P_device, P_size);
    hipMemcpy(P_device, P, P_size, hipMemcpyHostToDevice);
    hipMalloc(&P_device_target, P_size);
    hipMemcpy(P_device_target, P, P_size, hipMemcpyHostToDevice);
    float *Q_device, *Q_device_target;
    int Q_size = cols * n_factors * sizeof(float);
    hipMalloc(&Q_device, Q_size);
    hipMemcpy(Q_device, Q, Q_size, hipMemcpyHostToDevice);
    hipMalloc(&Q_device_target, Q_size);
    hipMemcpy(Q_device_target, Q, Q_size, hipMemcpyHostToDevice);

    // Create the errors - we would get this through the loss function
    float *errors = new float[matrix->nonzeros];
    for(int i = 0; i < matrix->nonzeros; ++i) {
        errors[i] = 1;
    }
    float *errors_device;
    hipMalloc(&errors_device, matrix->nonzeros * sizeof(float));
    hipMemcpy(errors_device, errors, matrix->nonzeros * sizeof(float), hipMemcpyHostToDevice);

    // Dimensions
    int n_threads = 32;
    dim3 dimBlock(n_threads);
    dim3 dimGrid(rows / n_threads + 1);
    sgd_update<<<dimGrid, dimBlock>>>(matrix->indptr, matrix->indices, P_device, Q_device, P_device_target, Q_device_target, n_factors, errors_device, rows, cols, learning_rate);
    std::swap(P_device, P_device_target);
    std::swap(Q_device, Q_device_target);

    // Copy updated P and Q back
    float *P_updated = new float[rows * n_factors];
    float *Q_updated = new float[cols * n_factors];
    hipMemcpy(P_updated, P_device, P_size, hipMemcpyDeviceToHost);
    hipMemcpy(Q_updated, Q_device, Q_size, hipMemcpyDeviceToHost);

    // For now, print the matrices
    cout << "Updated P: ";
    for(int i = 0; i < rows; ++i) {
        cout << P_updated[i] << " ";
    }
    cout << endl << "Updated Q: ";
    for(int i = 0; i < cols; ++i) {
        cout << Q_updated[i] << " ";
    }
    cout << endl;

    // Clean up
    hipFree(P_device);
    hipFree(P_device_target);
    hipFree(Q_device);
    hipFree(Q_device_target);
    hipFree(errors_device);
    delete matrix;
    delete P;
    delete P_updated;
    delete Q;
    delete Q_updated;
    delete errors;
}

int main() {
    test_sgd();
    hipDeviceSynchronize();
    return 0;
}