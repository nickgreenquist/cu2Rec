#include <fstream>

#include "config.h"

namespace config {
    bool Config::read_config(string file_path) {
        std::ifstream config_file(file_path);
        config_file >> cur_iterations >> total_iterations >> n_factors >> learning_rate >>
            seed >> P_reg >> Q_reg >> user_bias_reg >> item_bias_reg;
        config_file.close();
        return true;
    }

    bool Config::write_config(string file_path) {
        std::ofstream config_file(file_path);
        config_file << cur_iterations << " " << total_iterations << " " << n_factors << " " <<
        learning_rate << " " << seed << " " << P_reg << " " << Q_reg << " " << user_bias_reg <<
        " " << item_bias_reg << "\n";
        config_file.close();
        return true;
    }

    bool Config::set_cuda_variables() {
        hipMemcpyToSymbol(HIP_SYMBOL(config::cur_iterations), &cur_iterations, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(config::total_iterations), &total_iterations, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(config::n_factors), &n_factors, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(config::learning_rate), &learning_rate, sizeof(float));
        hipMemcpyToSymbol(HIP_SYMBOL(config::seed), &seed, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(config::P_reg), &P_reg, sizeof(float));
        hipMemcpyToSymbol(HIP_SYMBOL(config::Q_reg), &Q_reg, sizeof(float));
        hipMemcpyToSymbol(HIP_SYMBOL(config::user_bias_reg), &user_bias_reg, sizeof(float));
        hipMemcpyToSymbol(HIP_SYMBOL(config::item_bias_reg), &item_bias_reg, sizeof(float));

        hipError_t lastError;
        lastError = hipGetLastError();
        if(hipSuccess != lastError) {
            printf("ERROR: %s\n", hipGetErrorName(lastError));
            return false;
        }
        return true;
    }

    bool Config::get_cuda_variables() {
        hipMemcpyFromSymbol(&cur_iterations, HIP_SYMBOL(config::cur_iterations), sizeof(int));
        hipMemcpyFromSymbol(&total_iterations, HIP_SYMBOL(config::total_iterations), sizeof(int));
        hipMemcpyFromSymbol(&n_factors, HIP_SYMBOL(config::n_factors), sizeof(int));
        hipMemcpyFromSymbol(&learning_rate, HIP_SYMBOL(config::learning_rate), sizeof(float));
        hipMemcpyFromSymbol(&seed, HIP_SYMBOL(config::seed), sizeof(int));
        hipMemcpyFromSymbol(&P_reg, HIP_SYMBOL(config::P_reg), sizeof(float));
        hipMemcpyFromSymbol(&Q_reg, HIP_SYMBOL(config::Q_reg), sizeof(float));
        hipMemcpyFromSymbol(&user_bias_reg, HIP_SYMBOL(config::user_bias_reg), sizeof(float));
        hipMemcpyFromSymbol(&item_bias_reg, HIP_SYMBOL(config::item_bias_reg), sizeof(float));

        hipError_t lastError;
        lastError = hipGetLastError();
        if(hipSuccess != lastError) {
            printf("ERROR: %s\n", hipGetErrorName(lastError));
            return false;
        }
        return true;
    }
}
