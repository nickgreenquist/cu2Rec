#include <fstream>
#include "hip/hip_runtime.h"
#include "config.h"
#include "util.h"

namespace config {
    bool Config::read_config(std::string file_path) {
        std::ifstream config_file(file_path);
        config_file >> cur_iterations >> total_iterations >> n_factors >> learning_rate >>
            seed >> P_reg >> Q_reg >> user_bias_reg >> item_bias_reg;
        config_file.close();
        return true;
    }

    bool Config::write_config(std::string file_path) {
        std::ofstream config_file(file_path);
        config_file << cur_iterations << " " << total_iterations << " " << n_factors << " " <<
        learning_rate << " " << seed << " " << P_reg << " " << Q_reg << " " << user_bias_reg <<
        " " << item_bias_reg << "\n";
        config_file.close();
        return true;
    }

    bool Config::set_cuda_variables() {
        CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(config::cur_iterations), &cur_iterations, sizeof(int)));
        CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(config::total_iterations), &total_iterations, sizeof(int)));
        CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(config::n_factors), &n_factors, sizeof(int)));
        CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(config::learning_rate), &learning_rate, sizeof(float)));
        CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(config::seed), &seed, sizeof(int)));
        CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(config::P_reg), &P_reg, sizeof(float)));
        CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(config::Q_reg), &Q_reg, sizeof(float)));
        CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(config::user_bias_reg), &user_bias_reg, sizeof(float)));
        CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(config::item_bias_reg), &item_bias_reg, sizeof(float)));
        return true;
    }

    bool Config::get_cuda_variables() {
        CHECK_CUDA(hipMemcpyFromSymbol(&cur_iterations, HIP_SYMBOL(config::cur_iterations), sizeof(int)));
        CHECK_CUDA(hipMemcpyFromSymbol(&total_iterations, HIP_SYMBOL(config::total_iterations), sizeof(int)));
        CHECK_CUDA(hipMemcpyFromSymbol(&n_factors, HIP_SYMBOL(config::n_factors), sizeof(int)));
        CHECK_CUDA(hipMemcpyFromSymbol(&learning_rate, HIP_SYMBOL(config::learning_rate), sizeof(float)));
        CHECK_CUDA(hipMemcpyFromSymbol(&seed, HIP_SYMBOL(config::seed), sizeof(int)));
        CHECK_CUDA(hipMemcpyFromSymbol(&P_reg, HIP_SYMBOL(config::P_reg), sizeof(float)));
        CHECK_CUDA(hipMemcpyFromSymbol(&Q_reg, HIP_SYMBOL(config::Q_reg), sizeof(float)));
        CHECK_CUDA(hipMemcpyFromSymbol(&user_bias_reg, HIP_SYMBOL(config::user_bias_reg), sizeof(float)));
        CHECK_CUDA(hipMemcpyFromSymbol(&item_bias_reg, HIP_SYMBOL(config::item_bias_reg), sizeof(float)));
        return true;
    }

    void Config::print_config() {
        printf("Hyperparameters:\n");
        printf("total_iterations: %d\n", total_iterations);
        printf("n_factors: %d\n", n_factors);
        printf("learning_rate: %f\n", learning_rate);
        printf("P_reg: %f\n", P_reg);
        printf("Q_reg: %f\n", Q_reg);
        printf("user_bias_reg: %f\n", user_bias_reg);
        printf("item_bias_reg: %f\n", item_bias_reg);
        printf("is_train: %s\n", is_train?"true":"false");
        printf("n_threads: %d\n", n_threads);
        printf("check_error: %d\n", check_error);
        printf("patience: %f\n", patience);
        printf("learning_rate_decay: %f\n", learning_rate_decay);
    }
}
