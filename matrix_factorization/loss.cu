#include "hip/hip_runtime.h"
#include <stdexcept>
#include <sstream>
#include <iostream>     // std::cout
#include <math.h>       /* pow */

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "matrix.h"

#define index(i, j, N)  ((i)*(N)) + (j)
#define warp_size 32 //TODO: we need to get device props

using namespace cu2rec;

// PARALLEL
extern __shared__ float biases[];
__global__ void loss_kernel(int factors, int user_count, int item_count, const float * P, const float * Q, const int * indptr, 
                            const int * indices, const float * data, float * error, float * user_bias, float * item_bias, float global_bias) {
    float* s_user_bias = (float*)biases;
    float* s_item_bias = (float*)&s_user_bias[user_count];

    // use first warp to load in user_biases
    if(threadIdx.x < warp_size) {
        for(int i = 0; i < user_count; i += warp_size) {
            s_user_bias[i] = user_bias[i];
        }
    }
    // use second warp to load in item_biases
    if(threadIdx.x >= warp_size && threadIdx.x < 2*warp_size) {
        for(int i = 0; i < item_count; i += warp_size) {
            s_item_bias[i] = item_bias[i];
        }
    }
    // sync all threads before accessing any shared memory
    __syncthreads();
    
    // One thread per user
    int u = blockDim.x * blockIdx.x + threadIdx.x;
    if(u < user_count) {
        // get this user's factors into closer memory
        const float * p = &P[u * factors];
        const float ub = s_user_bias[u];

        for (int i = indptr[u]; i < indptr[u + 1]; ++i) {
            // get this item's factors
            int item_id = indices[i];
            const float * Qi = &Q[item_id * factors];

            // calculate predicted rating
            float pred = global_bias + ub + s_item_bias[item_id];
            for (int f = 0; f < factors; f++)
                pred += Qi[f]*p[f];

            // set the error value for this rating: rating - pred
            error[i] = data[i] - pred;
        }
    }
}

__global__ void total_loss_kernel(float *errors, float *losses, int n_errors, int current_iter, float discount) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    for(int i = n_errors / 2; i > 0; i >>= 1) {
        __syncthreads();
        if(x < i) {
            if(i == n_errors / 2) {
                // First iteration
                // Need to square the errors
                errors[x] = pow(errors[x], 2) + pow(errors[x + i], 2);
            } else {
                errors[x] += errors[x + i];
            }
        }
    }
    if(x == 0) {
        // Doing this atomic, in case we want to parallelize this calculation using streams
        atomicAdd(&losses[current_iter], discount * errors[0]);
    }
}

void calculate_loss_gpu(CudaDenseMatrix* P_d, CudaDenseMatrix* Q_d, int factors, int user_count, int item_count, int num_ratings, 
                        CudaCSRMatrix* matrix, float * error_d, float * user_bias,  float * item_bias, float global_bias) {
    int n_threads = 32;
    dim3 dimBlock(n_threads);
    dim3 dimGrid(user_count / n_threads + 1);
    float shared_mem_size = (user_count + item_count) * sizeof(float);
    loss_kernel<<<dimGrid, dimBlock, shared_mem_size>>>(
        factors, user_count, item_count, P_d->data, Q_d->data,
        matrix->indptr, matrix->indices, matrix->data, error_d,
        user_bias, item_bias, global_bias);
    hipError_t lastError = hipGetLastError();
    if(hipSuccess != lastError) {
        printf("ERROR: %s\n", hipGetErrorName(lastError));
    }
}
