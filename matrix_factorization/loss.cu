#include "hip/hip_runtime.h"
#include <stdexcept>
#include <sstream>
#include <iostream>     // std::cout
#include <math.h>       /* pow */

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "matrix.h"

#define index(i, j, N)  ((i)*(N)) + (j)

using namespace cu2rec;

// PARALLEL
__global__ void loss_kernel(int factors, int user_count, int item_count, const float * P, const float * Q, const int * indptr, 
                            const int * indices, const float * data, float * error, float * user_bias, float * item_bias, float global_bias) {
    // One thread per user
    int u = blockDim.x * blockIdx.x + threadIdx.x;
    if(u < user_count) {
        // get this user's factors
        const float * p = &P[u * factors];

        for (int i = indptr[u]; i < indptr[u + 1]; ++i) {
            // get this item's factors
            int item_id = indices[i];
            const float * Qi = &Q[item_id * factors];

            // calculate predicted rating
            float pred = global_bias + user_bias[u] + item_bias[item_id];
            for (int f = 0; f < factors; f++)
                pred += Qi[f]*p[f];

            // set the error value for this rating: rating - pred
            error[i] = data[i] - pred;
        }
    }
}

__global__ void total_loss_kernel(float *errors, float *losses, int n_errors, int current_iter, float discount) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    for(int i = n_errors / 2; i > 0; i >>= 1) {
        __syncthreads();
        if(x < i) {
            if(i == n_errors / 2) {
                // First iteration
                // Need to square the errors
                errors[x] = pow(errors[x], 2) + pow(errors[x + i], 2);
            } else {
                errors[x] += errors[x + i];
            }
        }
    }
    if(x == 0) {
        // Doing this atomic, in case we want to parallelize this calculation using streams
        atomicAdd(&losses[current_iter], discount * errors[0]);
    }
}

void calculate_loss_gpu(CudaDenseMatrix* P_d, CudaDenseMatrix* Q_d, int factors, int user_count, int item_count, int num_ratings, 
                        CudaCSRMatrix* matrix, float * error_d, float * user_bias,  float * item_bias, float global_bias) {
    int n_threads = 32;
    dim3 dimBlock(n_threads);
    dim3 dimGrid(user_count / n_threads + 1);
    loss_kernel<<<dimGrid, dimBlock>>>(
        factors, user_count, item_count, P_d->data, Q_d->data,
        matrix->indptr, matrix->indices, matrix->data, error_d,
        user_bias, item_bias, global_bias);
    hipError_t lastError = hipGetLastError();
    if(hipSuccess != lastError) {
        printf("ERROR: %s\n", hipGetErrorName(lastError));
    }
}

// SEQUENTIAL
float dot_product_sequential(const float *Qi, const float *p, int n) {
    float result = 0.0;
    for (int i = 0; i < n; i++)
        result += Qi[i]*p[i];
    return result;
}
float calculate_loss_sequential(int factors, int user_count, int item_count, const float * P, const float * Q, const int * indptr, const int * indices, const float * data) {
    float total_loss = 0;
    for(int u = 0; u < user_count; u++) {
        // get this user's factors
        float *p = new float[factors];
        for(int f = 0; f < factors; f++) {
            p[f] = P[index(u, f, factors)];
        }

        for (int i = indptr[u]; i < indptr[u + 1]; ++i) {
            // get this item's factors
            float *Qi = new float[factors];
            int item_id = indices[i];
            for(int f = 0; f < factors; f++) {
                Qi[f] = Q[index(item_id, f, factors)];
            }

            // update loss with this rating and prediction
            float rating = data[i];
            float pred = dot_product_sequential(Qi, p, factors);

            // std::cout << "Rating: " << rating << ", Pred: " << pred << "\n";

            float loss = pow(rating - pred, 2);
            total_loss += loss;

            delete [] Qi;
        }
        delete [] p;
    }
    return total_loss;
}
