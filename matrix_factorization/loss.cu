#include "hip/hip_runtime.h"
#include <stdexcept>
#include <sstream>
#include <iostream>     // std::cout
#include <math.h>       /* pow */
#include <tuple>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "matrix.h"
#include "util.h"

using namespace cu2rec;

// PARALLEL
__global__ void loss_kernel(int factors, int user_count, int item_count, const float * P, const float * Q, const int * indptr, 
                            const int * indices, const float * data, float * error, float * user_bias, float * item_bias, float global_bias) {
    
    // One thread per user
    int u = blockDim.x * blockIdx.x + threadIdx.x;
    if(u < user_count) {
        // get this user's factors into closer memory
        const float * p = &P[u * factors];
        const float ub = user_bias[u];

        for (int i = indptr[u]; i < indptr[u + 1]; ++i) {
            int item_id = indices[i];
            error[i] = data[i] - get_prediction(factors, p, &Q[item_id * factors], ub, item_bias[item_id], global_bias);
        }
    }
}

void calculate_loss_gpu(CudaDenseMatrix* P_d, CudaDenseMatrix* Q_d, config::Config* cfg, int user_count, int item_count, int num_ratings, 
                        CudaCSRMatrix* matrix, float * error_d, float * user_bias,  float * item_bias, float global_bias) {
    dim3 dimBlock(cfg->n_threads);
    dim3 dimGrid(user_count / cfg->n_threads + 1);
    loss_kernel<<<dimGrid, dimBlock>>>(
        cfg->n_factors, user_count, item_count, P_d->data, Q_d->data,
        matrix->indptr, matrix->indices, matrix->data, error_d,
        user_bias, item_bias, global_bias);
    CHECK_CUDA(hipGetLastError());
}

// Inspired by https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
// and https://devblogs.nvidia.com/using-shared-memory-cuda-cc/
// Fixes the problems related to data sizes
template <unsigned int block_size>
__global__ void total_loss_kernel(float *in_errors, float *out_errors, int n_errors, ErrorType error_type) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * block_size + tid;
    unsigned int grid_size = block_size * gridDim.x;
    sdata[tid] = 0;
    while (i < n_errors) {
        sdata[tid] += error_type == RMSE ? pow(in_errors[i], 2) : abs(in_errors[i]);
        i += grid_size;
    }
    __syncthreads();
    if (block_size >= 512) {
        if (tid < 256) {
            sdata[tid] += sdata[tid + 256];
        }
        __syncthreads();
    }
    if (block_size >= 256) {
        if (tid < 128) {
            sdata[tid] += sdata[tid + 128];
        }
        __syncthreads();
    }
    if (block_size >= 128) {
        if (tid < 64) {
            sdata[tid] += sdata[tid + 64];
        }
        __syncthreads();
    }
    if (block_size >= 64) {
        if (tid < 32) {
            sdata[tid] += sdata[tid + 32];
        }
        __syncthreads();
    }
    // if block_size is 1, compiler will complain about unneeded unsigned
    // int comparison (tid) to a value of 0
    if (!(block_size == 1) && tid < block_size / 2) {
        if (block_size >= 32) {
            sdata[tid] += sdata[tid + 16];
            __syncthreads();
        }
        if (block_size >= 16) {
            sdata[tid] += sdata[tid + 8];
            __syncthreads();
        }
        if (block_size >= 8) {
            sdata[tid] += sdata[tid + 4];
            __syncthreads();
        }
        if (block_size >= 4) {
            sdata[tid] += sdata[tid + 2];
            __syncthreads();
        }
        if (block_size >= 2) {
            sdata[tid] += sdata[tid + 1];
            __syncthreads();
        }
    }
    if (tid == 0) out_errors[blockIdx.x] = sdata[0];
}

std::tuple<float, float> get_error_metrics_cpu(float *errors, float *errors_device, int n_errors) {
    hipMemcpy(errors, errors_device, n_errors * sizeof(float), hipMemcpyDeviceToHost);
    float mae = 0.0;
    float rmse = 0.0;
    for(int k = 0; k <  n_errors; k++) {
        mae += abs(errors[k]);
        rmse += errors[k] * errors[k];
    }
    mae /= n_errors;
    rmse = sqrt(rmse / n_errors);
    return std::make_tuple(mae, rmse);
}

// Inspired by https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
// and https://devblogs.nvidia.com/using-shared-memory-cuda-cc/
// Fixes the problems related to data sizes
float calculate_error_metric_gpu(float *in_errors, float *out_errors, float *out_errors_host, int n_errors, int grid_size, int block_size, ErrorType error_type) {
    switch(block_size) {
        case 512:
            total_loss_kernel<512><<<grid_size, block_size, 512 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 256:
            total_loss_kernel<256><<<grid_size, block_size, 256 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 128:
            total_loss_kernel<128><<<grid_size, block_size, 128 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 64:
            total_loss_kernel< 64><<<grid_size, block_size,  64 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 32:
            total_loss_kernel< 32><<<grid_size, block_size,  32 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 16:
            total_loss_kernel< 16><<<grid_size, block_size,  16 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 8:
            total_loss_kernel<  8><<<grid_size, block_size,   8 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 4:
            total_loss_kernel<  4><<<grid_size, block_size,   4 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 2:
            total_loss_kernel<  2><<<grid_size, block_size,   2 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 1:
            total_loss_kernel<  1><<<grid_size, block_size,   1 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
    }
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipMemcpy(out_errors_host, out_errors, grid_size * sizeof(float), hipMemcpyDeviceToHost));
    float total = 0;
    for(int k = 0; k < grid_size; k++) {
        total += out_errors_host[k];
    }
    return error_type == RMSE ? sqrt(total / n_errors) : total / n_errors;
}

std::tuple<float, float> get_error_metrics_gpu(float *in_errors, float *out_errors, float *out_errors_host, int n_errors, int grid_size, int block_size) {
    float mae = calculate_error_metric_gpu(in_errors, out_errors, out_errors_host, n_errors, grid_size, block_size, MAE);
    float rmse = calculate_error_metric_gpu(in_errors, out_errors, out_errors_host, n_errors, grid_size, block_size, RMSE);
    return std::make_tuple(mae, rmse);
}
