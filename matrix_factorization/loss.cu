#include "hip/hip_runtime.h"
#include <stdexcept>
#include <sstream>
#include <iostream>     // std::cout
#include <math.h>       /* pow */

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "matrix.h"

#define index(i, j, N)  ((i)*(N)) + (j)

using namespace cu2rec;

// PARALLEL
__global__ void loss_kernel(int factors, int user_count, int item_count, const float * P, const float * Q, const int * indptr, 
                            const int * indices, const float * data, float * error, float * user_bias, float * item_bias, float global_bias) {
    // One thread per user
    int u = blockDim.x * blockIdx.x + threadIdx.x;
    if(u < user_count) {
        // get this user's factors
        const float * p = &P[u * factors];

        for (int i = indptr[u]; i < indptr[u + 1]; ++i) {
            // get this item's factors
            int item_id = indices[i];
            const float * Qi = &Q[item_id * factors];

            // calculate predicted rating
            float pred = global_bias + user_bias[u] + item_bias[item_id];
            for (int f = 0; f < factors; f++)
                pred += Qi[f]*p[f];

            // set the error value for this rating: rating - pred
            error[i] = data[i] - pred;
        }
    }
}

// Inspired by https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
// and https://devblogs.nvidia.com/using-shared-memory-cuda-cc/
// Fixes the problems related to data sizes
template <unsigned int block_size>
__global__ void total_loss_kernel(float *in_errors, float *out_errors, int n_errors) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * block_size + tid;
    unsigned int grid_size = block_size * gridDim.x;
    sdata[tid] = 0;
    while (i < n_errors) {
        sdata[tid] += pow(in_errors[i], 2);
        i += grid_size;
    }
    __syncthreads();
    if (block_size >= 512) {
        if (tid < 256) {
            sdata[tid] += sdata[tid + 256];
        }
        __syncthreads();
    }
    if (block_size >= 256) {
        if (tid < 128) {
            sdata[tid] += sdata[tid + 128];
        }
        __syncthreads();
    }
    if (block_size >= 128) {
        if (tid < 64) {
            sdata[tid] += sdata[tid + 64];
        }
        __syncthreads();
    }
    if (tid < block_size / 2) {
        if (block_size >= 64) sdata[tid] += sdata[tid + 32];
        if (block_size >= 32) sdata[tid] += sdata[tid + 16];
        if (block_size >= 16) sdata[tid] += sdata[tid + 8];
        if (block_size >= 8) sdata[tid] += sdata[tid + 4];
        if (block_size >= 4) sdata[tid] += sdata[tid + 2];
        if (block_size >= 2) sdata[tid] += sdata[tid + 1];
    }
    if (tid == 0) out_errors[blockIdx.x] = sdata[0];
}

void calculate_loss_gpu(CudaDenseMatrix* P_d, CudaDenseMatrix* Q_d, int factors, int user_count, int item_count, int num_ratings, 
                        CudaCSRMatrix* matrix, float * error_d, float * user_bias,  float * item_bias, float global_bias) {
    int n_threads = 32;
    dim3 dimBlock(n_threads);
    dim3 dimGrid(user_count / n_threads + 1);
    loss_kernel<<<dimGrid, dimBlock>>>(
        factors, user_count, item_count, P_d->data, Q_d->data,
        matrix->indptr, matrix->indices, matrix->data, error_d,
        user_bias, item_bias, global_bias);
    hipError_t lastError = hipGetLastError();
    if(hipSuccess != lastError) {
        printf("ERROR: %s\n", hipGetErrorName(lastError));
    }
}

// SEQUENTIAL
float dot_product_sequential(const float *Qi, const float *p, int n) {
    float result = 0.0;
    for (int i = 0; i < n; i++)
        result += Qi[i]*p[i];
    return result;
}
float calculate_loss_sequential(int factors, int user_count, int item_count, const float * P, const float * Q, const int * indptr, const int * indices, const float * data) {
    float total_loss = 0;
    for(int u = 0; u < user_count; u++) {
        // get this user's factors
        float *p = new float[factors];
        for(int f = 0; f < factors; f++) {
            p[f] = P[index(u, f, factors)];
        }

        for (int i = indptr[u]; i < indptr[u + 1]; ++i) {
            // get this item's factors
            float *Qi = new float[factors];
            int item_id = indices[i];
            for(int f = 0; f < factors; f++) {
                Qi[f] = Q[index(item_id, f, factors)];
            }

            // update loss with this rating and prediction
            float rating = data[i];
            float pred = dot_product_sequential(Qi, p, factors);

            // std::cout << "Rating: " << rating << ", Pred: " << pred << "\n";

            float loss = pow(rating - pred, 2);
            total_loss += loss;

            delete [] Qi;
        }
        delete [] p;
    }
    return total_loss;
}
