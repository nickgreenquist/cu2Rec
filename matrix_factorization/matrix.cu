// Credit: https://github.com/benfred/implicit
#include <stdexcept>
#include <sstream>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "matrix.h"

#define CHECK_CUDA(code) { checkCuda((code), __FILE__, __LINE__); }
inline void checkCuda(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::stringstream err;
        err << "Cuda Error: " << hipGetErrorString(code) << " (" << file << ":" << line << ")";
        throw std::runtime_error(err.str());
    }
}

namespace cu2rec {
    CudaDenseMatrix::CudaDenseMatrix(int rows, int cols, const float * host_data)
        : rows(rows), cols(cols) {
        CHECK_CUDA(hipMalloc(&data, rows * cols * sizeof(float)));
        if (host_data) {
            CHECK_CUDA(hipMemcpy(data, host_data, rows * cols * sizeof(float), hipMemcpyHostToDevice));
        }
    }

    void CudaDenseMatrix::to_host(float * out) const {
        CHECK_CUDA(hipMemcpy(out, data, rows * cols * sizeof(float), hipMemcpyDeviceToHost));
    }

    CudaDenseMatrix::~CudaDenseMatrix() {
        CHECK_CUDA(hipFree(data));
    }

    CudaCSRMatrix::CudaCSRMatrix(int rows, int cols, int nonzeros,
                                const int * indptr_, const int * indices_, const float * data_)
        : rows(rows), cols(cols), nonzeros(nonzeros) {

        CHECK_CUDA(hipMalloc(&indptr, (rows + 1) * sizeof(int)));
        CHECK_CUDA(hipMemcpy(indptr, indptr_, (rows + 1)*sizeof(int), hipMemcpyHostToDevice));

        CHECK_CUDA(hipMalloc(&indices, nonzeros * sizeof(int)));
        CHECK_CUDA(hipMemcpy(indices, indices_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

        CHECK_CUDA(hipMalloc(&data, nonzeros * sizeof(float)));
        CHECK_CUDA(hipMemcpy(data, data_, nonzeros * sizeof(int), hipMemcpyHostToDevice));
    }

    CudaCSRMatrix::~CudaCSRMatrix() {
        CHECK_CUDA(hipFree(indices));
        CHECK_CUDA(hipFree(indptr));
        CHECK_CUDA(hipFree(data));
    }
}  // namespace cu2rec
