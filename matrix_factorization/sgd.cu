#include "hip/hip_runtime.h"
#include <stdio.h>

#include "config.h"

/* To index element (i,j) of a 2D array stored as 1D */
#define index(i, j, N)  ((i)*(N)) + (j)
#define warp_size 32 //TODO: we need to get device props

/*************************/
/* CURAND INITIALIZATION */
/*************************/
__global__ void initCurand(hiprandState *state, unsigned long seed, int n_rows){
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if(x < n_rows && x < 1000) {
        hiprand_init(seed, x, 0, &state[x]);
    }
}

extern __shared__ float biases[];
__global__ void sgd_update(int *indptr, int *indices, float *P, float *Q, float *Q_target, 
                           float *errors, int n_rows, int n_cols, float *user_bias, float *item_bias,
                           float *item_bias_target, hiprandState *my_curandstate) {

    float* s_user_bias = (float*)biases;
    float* s_item_bias = (float*)&s_user_bias[n_rows];

    // use first warp to load in user_biases
    if(threadIdx.x < warp_size) {
        for(int i = 0; i < n_rows; i += warp_size) {
            s_user_bias[i] = user_bias[i];
        }
    }
    // use second warp to load in item_biases
    if(threadIdx.x >= warp_size && threadIdx.x < 2*warp_size) {
        for(int i = 0; i < n_cols; i += warp_size) {
            s_item_bias[i] = item_bias[i];
        }
    }
    // sync all threads before accessing any shared memory
    __syncthreads();

    // One thread per user
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if(x < n_rows) {
        
        // pick a random y_i
        int low = indptr[x];
        int high = indptr[x+1];
        float myrandf = hiprand_uniform(&my_curandstate[x % 1000]);
        myrandf *= (high - low + 0.999999);
        myrandf += low;
        int y_i = (int)truncf(myrandf);

        int y = indices[y_i];
        float error_y_i = errors[y_i];
        for(int f = 0; f < config::n_factors; ++f) {
            int p_index = index(x, f, config::n_factors);
            int q_index = index(y, f, config::n_factors);

            // Update P
            P[p_index] += config::learning_rate * (error_y_i * Q[q_index] - config::P_reg * P[p_index]);

            // Only update Q if train flag is true
            if(config::is_train) {
                Q_target[q_index] = Q[q_index] + config::learning_rate * (error_y_i * P[p_index] - config::Q_reg * Q[q_index]);
            }
        }

        // update biases
        user_bias[x] += config::learning_rate * (error_y_i - config::user_bias_reg * user_bias[x]);
        if(config::is_train) {
            item_bias_target[y] = item_bias[y] + config::learning_rate * (error_y_i - config::item_bias_reg * item_bias[y]);
        }

        // TODO: remove old loop over all items once we agree on one item per user SGD
        // Loop over all the ratings of the user
        // for(int y_i = indptr[x]; y_i < indptr[x + 1]; ++y_i) {
        // }
    }
}