#include "hip/hip_runtime.h"
#include <stdio.h>

#include "config.h"

/* To index element (i,j) of a 2D array stored as 1D */
#define index(i, j, N)  ((i)*(N)) + (j)

/*************************/
/* CURAND INITIALIZATION */
/*************************/
__global__ void initCurand(hiprandState *state, unsigned long seed, int n_rows){
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if(x < n_rows && x < 1000) {
        hiprand_init(seed, x, 0, &state[x]);
    }
}

__global__ void sgd_update(int *indptr, int *indices, float *P, float *Q, float *P_target, float *Q_target, 
                           float *errors, int n_rows, int n_cols, float *user_bias, float *item_bias,
                           float *user_bias_target, float *item_bias_target, hiprandState *my_curandstate) {
    // One thread per user
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if(x < n_rows) {
        
        // pick a random y_i
        int low = indptr[x];
        int high = indptr[x+1];
        float myrandf = hiprand_uniform(&my_curandstate[x % 1000]);
        myrandf *= (high - low + 0.999999);
        myrandf += low;
        int y_i = (int)truncf(myrandf);

        int y = indices[y_i];
        for(int f = 0; f < config::n_factors; ++f) {
            int p_index = index(x, f, config::n_factors);
            int q_index = index(y, f, config::n_factors);

            // Update P
            P_target[p_index] = P[p_index] + config::learning_rate * (errors[y_i] * Q[q_index] - config::P_reg * P[p_index]);

            // Only update Q if train flag is true
            if(config::is_train) {
                Q_target[q_index] = Q[q_index] + config::learning_rate * (errors[y_i] * P[p_index] - config::Q_reg * Q[q_index]);
            }
        }

        // update biases
        user_bias_target[x] = user_bias[x] + config::learning_rate * (errors[y_i] - config::user_bias_reg * user_bias[x]);
        if(config::is_train) {
            item_bias_target[y] = item_bias[y] + config::learning_rate * (errors[y_i] - config::item_bias_reg * item_bias[y]);
        }

        // TODO: remove old loop over all items once we agree on one item per user SGD
        // Loop over all the ratings of the user
        // for(int y_i = indptr[x]; y_i < indptr[x + 1]; ++y_i) {
        // }
    }
}