#include "hip/hip_runtime.h"
#include <stdio.h>

#include "matrix.h"

/* To index element (i,j) of a 2D array stored as 1D */
#define index(i, j, N)  ((i)*(N)) + (j)

__global__ void sgd_update(int *indptr, int *indices, float *P, float *Q, float *P_target, float *Q_target, int n_factors, 
                           float *errors, int n_rows, int n_cols, float learning_rate, float *user_bias, float *item_bias,
                           float *user_bias_target, float *item_bias_target, float user_bias_reg, float item_bias_reg) {
    // One thread per user
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if(x < n_rows) {
        // Loop over all the ratings of the user
        for(int y_i = indptr[x]; y_i < indptr[x + 1]; ++y_i) {
            int y = indices[y_i];
            for(int f = 0; f < n_factors; ++f) {
                int p_index = index(x, f, n_factors);
                int q_index = index(y, f, n_factors);
                // printf("User %d item %d updating P %d (%d, %d) and Q %d (%d, %d)\n", x, y, p_index, x, f, q_index, f, y);

                // Update user and item biases
                float ub_update = learning_rate * (errors[y_i] - user_bias_reg * user_bias[x]);
                user_bias_target[x] += ub_update;
                float ib_update = learning_rate * (errors[y_i] - item_bias_reg * item_bias[y]);
                atomicAdd(&item_bias_target[y], ib_update);

                // Update latent factors
                float p_update = learning_rate * errors[y_i] * Q[q_index];
                P_target[p_index] += p_update;
                float q_update = learning_rate * errors[y_i] * P[p_index];
                atomicAdd(&Q_target[q_index], q_update);
            }
        }
    }
}