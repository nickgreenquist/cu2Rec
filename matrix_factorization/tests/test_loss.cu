#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>       /* pow */
#include <time.h>
#include <tuple>
#include <vector>

#include "../util.h"
#include "../matrix.h"
#include "../loss.h"

#define index(i, j, N)  ((i)*(N)) + (j)

using namespace std;

string filename = "../../data/test/test_ratings.csv";
int factors = 2;

void test_loss() {
    int rows, cols;
    float global_bias;
    vector<Rating> ratings = readCSV(filename, &rows, &cols, &global_bias);

    // set global_bias to 1.0 for easier testing
    global_bias = 1.0;

    // Create Sparse Matrix in Device memory
    cu2rec::CudaCSRMatrix* matrix = createSparseMatrix(&ratings, rows, cols);

    // create temp P and Q
    int user_count = rows;
    int item_count = cols;
    float *P = new float[user_count * factors];
    float *Q = new float[item_count * factors];
    for(int u = 0; u < user_count; u++) {
        for(int f = 0; f < factors; f++) {
            P[index(u, f, factors)] = 1.0;
        }
    }
    for(int i = 0; i < item_count; i++) {
        for(int f = 0; f < factors; f++) {
            Q[index(i, f, factors)] = 1.0;
        }
    }

    // make copy of error array
    float* error = new float[ratings.size()];
    float* error_d;
    hipMalloc((void **) &error_d, ratings.size() * sizeof(float));

    // create user and item bias arrays
    float *user_bias = new float[user_count];
    for(int u = 0; u < user_count; u++) {
        user_bias[u] = 1.0;
    }
    float *user_bias_device;
    hipMalloc((void **) &user_bias_device, user_count * sizeof(float));
    hipMemcpy(user_bias_device, user_bias, user_count * sizeof(float), hipMemcpyHostToDevice);

    float *item_bias = new float[item_count];
    for(int i = 0; i < item_count; i++) {
        item_bias[i] = 1.0;
    }
    float *item_bias_device;
    hipMalloc((void **) &item_bias_device, item_count * sizeof(float));
    hipMemcpy(item_bias_device, item_bias, item_count * sizeof(float), hipMemcpyHostToDevice);
    
    // Turn P and Q into CudaDenseMatrices on GPU and calculate the loss using GPU
    CudaDenseMatrix* P_d = new CudaDenseMatrix(user_count, factors, P);
    CudaDenseMatrix* Q_d = new CudaDenseMatrix(item_count, factors, Q);
    calculate_loss_gpu(P_d, Q_d, factors, user_count, item_count, ratings.size(), matrix, error_d, user_bias_device, item_bias_device, global_bias);

    // move array of errors back to host
    hipMemcpy(error, error_d, ratings.size() * sizeof(float), hipMemcpyDeviceToHost);

    float loss = 0.0;
    for(int i = 0; i < ratings.size(); i++) {
        loss += pow(error[i], 2);
    }

    cout << "\nLoss: " << loss << "\n";
    assert(loss == 74.0);

    //free memory
    delete matrix;
    delete [] P;
    delete [] Q;
    delete [] error;
    delete [] user_bias;
    hipFree(error_d);
    hipFree(user_bias_device);
    hipFree(item_bias_device);
}

void test_total_loss() {
    vector<int> problem_sizes = { 1, 33, 1<<10, 1<<16 };
    vector<int> grid_sizes = { 1, 20, 1000 };
    vector<int> block_sizes = { 1, 16, 64 };
    for (std::vector<int>::iterator i = problem_sizes.begin(); i != problem_sizes.end(); ++i) {
        for (std::vector<int>::iterator j = grid_sizes.begin(); j != grid_sizes.end(); ++j) {
            for (std::vector<int>::iterator k = block_sizes.begin(); k != block_sizes.end(); ++k) {
                // Set up experiment sizes
                int problem_size = (*i);
                int grid_size = (*j);
                int block_size = (*k);

                // Input and output arrays
                float *in_errors = new float[problem_size];
                for (int i = 0; i < problem_size; ++i) {
                    in_errors[i] = 1.0;
                }
                float *in_errors_device;
                hipMalloc(&in_errors_device, problem_size * sizeof(float));
                hipMemcpy(in_errors_device, in_errors, problem_size * sizeof(float), hipMemcpyHostToDevice);

                float *out_errors = new float[grid_size];
                float *out_errors_device;
                hipMalloc(&out_errors_device, grid_size * sizeof(float));

                // Call the kernel
                float mae, rmse;
                std::tie(mae, rmse) = get_error_metrics_gpu(in_errors_device, out_errors_device, out_errors, problem_size, grid_size, block_size);
                
                // Since all errors are 1.0, we expect the RMSE and MAE to be 1
                // This makes sure the kernel covers all problem_size elements
                assert(mae == 1);
                assert(rmse == 1);

                hipFree(in_errors_device);
                hipFree(out_errors_device);
                delete [] in_errors;
                delete [] out_errors;
            }
        }
    }
}

int main() {
    cout << "Testing Parallel Loss Function on test ratings...";
    test_loss();
    cout << "PASSED\n";

    cout << "Testing calculation of total loss...";
    test_total_loss();
    cout << "PASSED\n";

    return 0;
}
