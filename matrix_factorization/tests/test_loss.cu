#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>       /* pow */
#include <time.h>

#include "../util.h"
#include "../matrix.h"
#include "../loss.h"

#define index(i, j, N)  ((i)*(N)) + (j)

using namespace std;

string filename = "../../data/test/test_ratings.csv";
int factors = 2;

void test_loss() {
    int rows, cols;
    float global_bias;
    vector<Rating> ratings = readCSV(filename, &rows, &cols, &global_bias);

    // set global_bias to 1.0 for easier testing
    global_bias = 1.0;

    // Create Sparse Matrix in Device memory
    cu2rec::CudaCSRMatrix* matrix = createSparseMatrix(&ratings, rows, cols);

    // create temp P and Q
    int user_count = rows;
    int item_count = cols;
    float *P = new float[user_count * factors];
    float *Q = new float[item_count * factors];
    for(int u = 0; u < user_count; u++) {
        for(int f = 0; f < factors; f++) {
            P[index(u, f, factors)] = 1.0;
        }
    }
    for(int i = 0; i < item_count; i++) {
        for(int f = 0; f < factors; f++) {
            Q[index(i, f, factors)] = 1.0;
        }
    }

    // make copy of error array
    float* error = new float[ratings.size()];
    float* error_d;
    hipMalloc((void **) &error_d, ratings.size() * sizeof(float));

    // create user and item bias arrays
    float *user_bias = new float[user_count];
    for(int u = 0; u < user_count; u++) {
        user_bias[u] = 1.0;
    }
    float *user_bias_device;
    hipMalloc((void **) &user_bias_device, user_count * sizeof(float));
    hipMemcpy(user_bias_device, user_bias, user_count * sizeof(float), hipMemcpyHostToDevice);

    float *item_bias = new float[item_count];
    for(int i = 0; i < item_count; i++) {
        item_bias[i] = 1.0;
    }
    float *item_bias_device;
    hipMalloc((void **) &item_bias_device, item_count * sizeof(float));
    hipMemcpy(item_bias_device, item_bias, item_count * sizeof(float), hipMemcpyHostToDevice);
    
    // Turn P and Q into CudaDenseMatrices on GPU and calculate the loss using GPU
    CudaDenseMatrix* P_d = new CudaDenseMatrix(user_count, factors, P);
    CudaDenseMatrix* Q_d = new CudaDenseMatrix(item_count, factors, Q);
    calculate_loss_gpu(P_d, Q_d, factors, user_count, item_count, ratings.size(), matrix, error_d, user_bias_device, item_bias_device, global_bias);

    // move array of errors back to host
    hipMemcpy(error, error_d, ratings.size() * sizeof(float), hipMemcpyDeviceToHost);

    float loss = 0.0;
    for(int i = 0; i < ratings.size(); i++) {
        loss += pow(error[i], 2);
    }

    cout << "\nLoss: " << loss << "\n";
    assert(loss == 74.0);

    //free memory
    delete matrix;
    delete [] P;
    delete [] Q;
    delete [] error;
    delete [] user_bias;
    hipFree(error_d);
    hipFree(user_bias_device);
    hipFree(item_bias_device);
}

int main() {
    cout << "Testing Parallel Loss Function on test ratings...";
    test_loss();
    cout << "PASSED\n";

    return 0;
}
