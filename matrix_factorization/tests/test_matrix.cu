#include "hip/hip_runtime.h"
#include <iostream>     // std::cout
#include <fstream>      // std::ifstream
#include <algorithm>
#include <vector>
#include <string>
#include <assert.h>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "../matrix.h"
#include "../utils.cuh"

using namespace std;

__global__ void dot_kernel(float* a, float* b, float* c) {
    // for some reason, this kernel has to be called len(a) times for c to get the full dot product????
    *c = cu2rec::dot(a,b);
}

void test_dot() {
    vector<float> va = {1,2,3,4,5};
    vector<float> vb = {5,4,3,2,1};
    int size = va.size() * sizeof(float);

    float* a, *b, *c;
    a = (float *)malloc(size);
    b = (float *)malloc(size);
    c = (float *)malloc(sizeof(float));

    // Convert vector to float array
    for(int i = 0; i < va.size(); i++) {
        a[i] = va.at(i);
        b[i] = vb.at(i);
    }

    // Allocate and copy memory to device arrays
    float* a_d, *b_d, *c_d;

    hipMalloc((void **)&a_d, size);
    hipMalloc((void **)&b_d, size);
    hipMalloc((void **)&c_d, sizeof(float));

    hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, size, hipMemcpyHostToDevice);
    hipMemcpy(c_d, c, sizeof(float), hipMemcpyHostToDevice);

    dot_kernel<<<1, va.size()>>>(a_d,b_d,c_d);

    hipMemcpy(c, c_d, sizeof(float), hipMemcpyDeviceToHost);
    
    assert(*c == 35);

    hipFree(a_d); hipFree(b_d); hipFree(c_d);
}

int main() {
    cout << "Testing dot() function returns correct result...";
    test_dot();
    cout << "PASSED\n";

    return 0;
}