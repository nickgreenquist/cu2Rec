#include "hip/hip_runtime.h"
#include <random>
#include <cmath>
#include <time.h>
#include <tuple>
#include <limits>

#include "config.h"
#include "loss.h"
#include "matrix.h"
#include "sgd.h"
#include "util.h"

using namespace cu2rec;

void train(CudaCSRMatrix* train_matrix, CudaCSRMatrix* test_matrix, config::Config* cfg, float **P_ptr, float **Q_ptr, float *Q, float **losses_ptr,
           float **user_bias_ptr, float **item_bias_ptr, float *item_bias, float global_bias) {
    int user_count = train_matrix->rows;
    int item_count = train_matrix->cols;
    cfg->set_cuda_variables();

    // Initialize P, Q has already been initialized
    float *P = initialize_normal_array(user_count * cfg->n_factors, cfg->n_factors);
    float *losses = new float[cfg->total_iterations];
    *P_ptr = P;
    *losses_ptr = losses;

    // Copy P and Q to device memory
    CudaDenseMatrix* P_device = new CudaDenseMatrix(user_count, cfg->n_factors, P);
    CudaDenseMatrix* Q_device = new CudaDenseMatrix(item_count, cfg->n_factors, Q);
    CudaDenseMatrix* P_device_target = new CudaDenseMatrix(user_count, cfg->n_factors, P);
    CudaDenseMatrix* Q_device_target = new CudaDenseMatrix(item_count, cfg->n_factors, Q);

    // Create the errors
    float *errors = new float[train_matrix->nonzeros];
    float *errors_device;
    CHECK_CUDA(hipMalloc(&errors_device, train_matrix->nonzeros * sizeof(float)));

    float *errors_test = new float[test_matrix->nonzeros];
    float *errors_test_device;
    CHECK_CUDA(hipMalloc(&errors_test_device, test_matrix->nonzeros * sizeof(float)));

    // Create the bias array
    float *user_bias = initialize_normal_array(user_count, cfg->n_factors);
    *user_bias_ptr = user_bias;
    
    float *user_bias_device;
    CHECK_CUDA(hipMalloc(&user_bias_device, user_count * sizeof(float)));
    CHECK_CUDA(hipMemcpy(user_bias_device, user_bias, user_count * sizeof(float), hipMemcpyHostToDevice));

    float *item_bias_device;
    CHECK_CUDA(hipMalloc(&item_bias_device, item_count * sizeof(float)));
    CHECK_CUDA(hipMemcpy(item_bias_device, item_bias, item_count * sizeof(float), hipMemcpyHostToDevice));

    // Create bias targets
    float *user_bias_target, *item_bias_target;
    CHECK_CUDA(hipMalloc(&user_bias_target, user_count * sizeof(float)));
    CHECK_CUDA(hipMemcpy(user_bias_target, user_bias, user_count * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMalloc(&item_bias_target, item_count * sizeof(float)));
    CHECK_CUDA(hipMemcpy(item_bias_target, item_bias, item_count * sizeof(float), hipMemcpyHostToDevice));

    // Dimensions
    dim3 dim_block(cfg->n_threads);
    dim3 dim_grid_sgd(user_count / cfg->n_threads + 1);
    dim3 dim_grid_loss(256); // TODO: figure out a way to use a config value for this
    dim3 dim_block_loss(2 * cfg->n_threads);
    dim3 dim_grid_P_reg_loss(P_device->rows * P_device->cols / cfg->n_threads + 1);
    dim3 dim_grid_Q_reg_loss(Q_device->rows * Q_device->cols / cfg->n_threads + 1);
    dim3 dim_grid_user_bias_reg_loss(user_count / cfg->n_threads + 1);
    dim3 dim_grid_item_bias_reg_loss(item_count / cfg->n_threads + 1);

    // Create loss per block
    double *block_errors_host = new double[dim_grid_loss.x];
    double *block_errors_device;
    CHECK_CUDA(hipMalloc(&block_errors_device, dim_grid_loss.x * sizeof(double)));
    CHECK_CUDA(hipMemset(block_errors_device, 0, dim_grid_loss.x * sizeof(double)));

    // Create hiprand state
    hiprandState *d_state;
    CHECK_CUDA(hipMalloc(&d_state, user_count * sizeof(hiprandState)));
    // Set up random state using iteration as seed
    initCurand<<<dim_grid_sgd, dim_block>>>(d_state, cfg->seed, user_count);
    CHECK_CUDA(hipGetLastError());

    // to measure time taken by a specific part of the code 
    double time_taken;
    clock_t start, end;

    // Adaptive learning rate setup
    float train_rmse, train_mae, validation_rmse, validation_mae, last_validation_rmse;
    validation_rmse = validation_mae = std::numeric_limits<float>::max();
    int current_patience = cfg->patience;

    // Training loop
    start = clock();
    for (int i = 0; i < cfg->total_iterations; ++i) {

        // Run single iteration of SGD
        sgd_update<<<dim_grid_sgd, dim_block>>>(train_matrix->indptr, train_matrix->indices, train_matrix->data, P_device->data, Q_device->data, 
                                                Q_device_target->data, user_count, user_bias_device, item_bias_device, item_bias_target, d_state,
                                                global_bias);
        CHECK_CUDA(hipGetLastError());

        // Calculate total loss first, last, and every check_error iterations
        if((i + 1) % cfg->check_error == 0 || i == 0 || (i + 1) % cfg->total_iterations == 0) {

            // Calculate error on train ratings
            calculate_loss_gpu(P_device, Q_device, cfg, user_count, item_count, train_matrix->nonzeros, train_matrix,
                               errors_device, user_bias_device, item_bias_device, global_bias);

            // Calculate error on test ratings
            calculate_loss_gpu(P_device, Q_device, cfg, test_matrix->rows, test_matrix->cols, test_matrix->nonzeros, test_matrix,
                               errors_test_device, user_bias_device, item_bias_device, global_bias);

            // save previous metrics
            last_validation_rmse = validation_rmse;

            // TODO add this as a param to train function
            bool use_gpu = true;
            if(use_gpu) {
                std::tie(train_mae, train_rmse) = get_error_metrics_gpu(errors_device, block_errors_device, block_errors_host, train_matrix->nonzeros, dim_grid_loss.x, dim_block_loss.x);
                printf("TRAIN: Iteration %d GPU MAE: %f RMSE: %f\n", i + 1, train_mae, train_rmse);
                std::tie(validation_mae, validation_rmse) = get_error_metrics_gpu(errors_test_device, block_errors_device, block_errors_host, test_matrix->nonzeros, dim_grid_loss.x, dim_block_loss.x);
                printf("TEST: Iteration %d GPU MAE: %f RMSE: %f\n", i + 1, validation_mae, validation_rmse);
            } else {
                std::tie(train_mae, train_rmse) = get_error_metrics_cpu(errors, errors_device, train_matrix->nonzeros);
                printf("TRAIN: Iteration %d MAE: %f RMSE: %f\n", i + 1, train_mae, train_rmse);
                std::tie(validation_mae, validation_rmse) = get_error_metrics_cpu(errors_test, errors_test_device, test_matrix->nonzeros);
                printf("TEST: Iteration %d MAE: %f RMSE: %f\n", i + 1, validation_mae, validation_rmse);
            }

            // Update learning rate if needed
            if(last_validation_rmse < validation_rmse) {
                current_patience--;
            }
            if(current_patience <= 0) {
                current_patience = cfg->patience;
                cfg->learning_rate *= cfg->learning_rate_decay;
                cfg->set_cuda_variables();

                printf("New Learning Rate: %f\n: ", cfg->learning_rate);
            }

            // TODO: Do we still need to store this?
            losses[i] = validation_rmse;
        }

        CHECK_CUDA(hipGetLastError());

        // Swap item related components
        swap(Q_device, Q_device_target);
        swap(item_bias_device, item_bias_target);

        cfg->cur_iterations += 1;
    }
    CHECK_CUDA(hipDeviceSynchronize());
    end = clock();

    // Output time taken
    time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;   
    printf("Time taken for %d of iterations is %lf\n", cfg->total_iterations, time_taken);

    // Copy updated P and Q back
    P_device->to_host(P);
    Q_device->to_host(Q);

    // Copy updated bias arrays back
    CHECK_CUDA(hipMemcpy(user_bias, user_bias_device, user_count * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(item_bias, item_bias_device, item_count * sizeof(float), hipMemcpyDeviceToHost));

    // Free memory
    CHECK_CUDA(hipFree(errors_device));
    CHECK_CUDA(hipFree(errors_test_device));
    CHECK_CUDA(hipFree(block_errors_device));
    CHECK_CUDA(hipFree(user_bias_device));
    CHECK_CUDA(hipFree(item_bias_device));
    CHECK_CUDA(hipFree(user_bias_target));
    CHECK_CUDA(hipFree(item_bias_target));
    CHECK_CUDA(hipFree(d_state));
    delete P_device;
    delete P_device_target;
    delete Q_device;
    delete Q_device_target;
    delete [] errors;
    delete [] errors_test;
    delete [] block_errors_host;
}

void train(CudaCSRMatrix* train_matrix, CudaCSRMatrix* test_matrix, config::Config* cfg, float **P_ptr, float **Q_ptr, float **losses_ptr,
           float **user_bias_ptr, float **item_bias_ptr, float global_bias) {
    int item_count = train_matrix->cols;
    // Initialize for regular training
    float *Q = initialize_normal_array(item_count * cfg->n_factors, cfg->n_factors);
    float *item_bias = initialize_normal_array(item_count, cfg->n_factors);
    *Q_ptr = Q;
    *item_bias_ptr = item_bias;
    train(train_matrix, test_matrix, cfg, P_ptr, Q_ptr, Q, losses_ptr, user_bias_ptr, item_bias_ptr, item_bias, global_bias);
}