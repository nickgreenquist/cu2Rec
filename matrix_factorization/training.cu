#include "hip/hip_runtime.h"
#include <random>
#include <cmath>
#include <time.h>
#include <tuple>

#include "config.h"
#include "loss.h"
#include "matrix.h"
#include "sgd.h"
#include "util.h"

#define index(i, j, N)  ((i)*(N)) + (j)

using namespace cu2rec;

void train(CudaCSRMatrix* train_matrix, CudaCSRMatrix* test_matrix, config::Config* cfg, float **P_ptr, float **Q_ptr, float *Q, float **losses_ptr,
           float **user_bias_ptr, float **item_bias_ptr, float *item_bias, float global_bias) {
    int user_count = train_matrix->rows;
    int item_count = train_matrix->cols;
    cfg->set_cuda_variables();

    // Initialize P, Q has already been initialized
    float *P = initialize_normal_array(user_count * cfg->n_factors, cfg->n_factors);
    float *losses = new float[cfg->total_iterations];
    *P_ptr = P;
    *losses_ptr = losses;

    // Copy P and Q to device memory
    CudaDenseMatrix* P_device = new CudaDenseMatrix(user_count, cfg->n_factors, P);
    CudaDenseMatrix* Q_device = new CudaDenseMatrix(item_count, cfg->n_factors, Q);
    CudaDenseMatrix* P_device_target = new CudaDenseMatrix(user_count, cfg->n_factors, P);
    CudaDenseMatrix* Q_device_target = new CudaDenseMatrix(item_count, cfg->n_factors, Q);

    // Create the errors
    float *errors = new float[train_matrix->nonzeros];
    float *errors_device;
    CHECK_CUDA(hipMalloc(&errors_device, train_matrix->nonzeros * sizeof(float)));

    float *errors_test = new float[test_matrix->nonzeros];
    float *errors_test_device;
    CHECK_CUDA(hipMalloc(&errors_test_device, test_matrix->nonzeros * sizeof(float)));

    // Create the bias array
    float *user_bias = initialize_normal_array(user_count, cfg->n_factors);
    *user_bias_ptr = user_bias;
    
    float *user_bias_device;
    CHECK_CUDA(hipMalloc(&user_bias_device, user_count * sizeof(float)));
    CHECK_CUDA(hipMemcpy(user_bias_device, user_bias, user_count * sizeof(float), hipMemcpyHostToDevice));

    float *item_bias_device;
    CHECK_CUDA(hipMalloc(&item_bias_device, item_count * sizeof(float)));
    CHECK_CUDA(hipMemcpy(item_bias_device, item_bias, item_count * sizeof(float), hipMemcpyHostToDevice));

    // Create bias targets
    float *user_bias_target, *item_bias_target;
    CHECK_CUDA(hipMalloc(&user_bias_target, user_count * sizeof(float)));
    CHECK_CUDA(hipMemcpy(user_bias_target, user_bias, user_count * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMalloc(&item_bias_target, item_count * sizeof(float)));
    CHECK_CUDA(hipMemcpy(item_bias_target, item_bias, item_count * sizeof(float), hipMemcpyHostToDevice));

    // Dimensions
    int minibatch_size = 1<<14;
    if(minibatch_size > user_count) {
        minibatch_size = user_count + 32;
    }
    int n_threads = 32;
    dim3 dim_block(n_threads);
    dim3 dim_grid_sgd(user_count / n_threads + 1);
    dim3 dim_grid_loss(256);
    dim3 dim_block_loss(64); // Must be 2^0 to 2^9
    dim3 dim_grid_P_reg_loss(P_device->rows * P_device->cols / n_threads + 1);
    dim3 dim_grid_Q_reg_loss(Q_device->rows * Q_device->cols / n_threads + 1);
    dim3 dim_grid_user_bias_reg_loss(user_count / n_threads + 1);
    dim3 dim_grid_item_bias_reg_loss(item_count / n_threads + 1);

    // Create loss per block
    float *block_errors_host = new float[dim_grid_loss.x];
    float *block_errors_device;
    CHECK_CUDA(hipMalloc(&block_errors_device, dim_grid_loss.x * sizeof(float)));
    CHECK_CUDA(hipMemset(block_errors_device, 0, dim_grid_loss.x * sizeof(float)));

    // Create hiprand state
    hiprandState *d_state;
    CHECK_CUDA(hipMalloc(&d_state, user_count * sizeof(hiprandState)));
    // Set up random state using iteration as seed
    initCurand<<<dim_grid_sgd, dim_block>>>(d_state, cfg->seed, user_count);
    CHECK_CUDA(hipGetLastError());

    // to measure time taken by a specific part of the code 
    double time_taken, time_taken_loss;
    clock_t start, end, start_loss, end_loss;

    int start_user = 0;
    // Training loop
    start = clock();
    for (int i = 0; i < cfg->total_iterations; ++i) {

        // Run single iteration of SGD
        sgd_update<<<minibatch_size/dim_block.x, dim_block>>>(train_matrix->indptr, train_matrix->indices, train_matrix->data, P_device->data, Q_device->data, 
                                                Q_device_target->data, user_count, user_bias_device, item_bias_device, item_bias_target, d_state,
                                                global_bias, start_user);

        start_user = (minibatch_size == user_count + 32) ? 0 : (start_user + minibatch_size) % user_count;
        CHECK_CUDA(hipGetLastError());

        // Calculate total loss periodically to check for improving loss
        // if((i + 1) % cfg->total_iterations == 0 || i == 0) {
        if((i + 1) % 100 == 0 || i == 0) {
            start_loss = clock();

            // Calculate error on train ratings
            calculate_loss_gpu(P_device, Q_device, cfg->n_factors, user_count, item_count, train_matrix->nonzeros, train_matrix,
                               errors_device, user_bias_device, item_bias_device, global_bias);

            // Calculate error on test ratings
            calculate_loss_gpu(P_device, Q_device, cfg->n_factors, test_matrix->rows, test_matrix->cols, test_matrix->nonzeros, test_matrix,
                               errors_test_device, user_bias_device, item_bias_device, global_bias);

            // TODO add this as a param to train function
            bool use_gpu = true;
            float mae, rmse;
            if(use_gpu) {
                std::tie(mae, rmse) = get_error_metrics_gpu(errors_device, block_errors_device, block_errors_host, train_matrix->nonzeros, dim_grid_loss.x, dim_block_loss.x);
                printf("TRAIN: Iteration %d GPU MAE: %f RMSE: %f\n", i + 1, mae, rmse);
                std::tie(mae, rmse) = get_error_metrics_gpu(errors_test_device, block_errors_device, block_errors_host, test_matrix->nonzeros, dim_grid_loss.x, dim_block_loss.x);
                printf("TEST: Iteration %d GPU MAE: %f RMSE: %f\n", i + 1, mae, rmse);
            } else {
                std::tie(mae, rmse) = get_error_metrics_cpu(errors, errors_device, train_matrix->nonzeros);
                printf("TRAIN: Iteration %d MAE: %f RMSE: %f\n", i + 1, mae, rmse);
                std::tie(mae, rmse) = get_error_metrics_cpu(errors_test, errors_test_device, test_matrix->nonzeros);
                printf("TEST: Iteration %d MAE: %f RMSE: %f\n", i + 1, mae, rmse);
            }
            losses[i] = rmse;

            end_loss = clock();
            time_taken_loss = ((double)(end_loss - start_loss))/ CLOCKS_PER_SEC;   
            printf("Time taken to calculate total loss is %lf\n\n", time_taken_loss);
        }
        // TODO move these to loss.cu
        // if(cfg->P_reg > 0)
        //     total_loss_kernel<<<dim_grid_P_reg_loss, dim_block>>>(P_device->data, losses_device, P_device->rows * P_device->cols, i, cfg->P_reg);
        // if(cfg->Q_reg > 0)
        //     total_loss_kernel<<<dim_grid_Q_reg_loss, dim_block>>>(Q_device->data, losses_device, Q_device->rows * Q_device->cols, i, cfg->Q_reg);
        // if(cfg->user_bias_reg > 0)
        //     total_loss_kernel<<<dim_grid_user_bias_reg_loss, dim_block>>>(user_bias_device, losses_device, user_count, i, cfg->user_bias_reg);
        // if(cfg->item_bias_reg > 0)
        //     total_loss_kernel<<<dim_grid_item_bias_reg_loss, dim_block>>>(item_bias_device, losses_device, item_count, i, cfg->item_bias_reg);

        CHECK_CUDA(hipGetLastError());

        // Swap item related components
        swap(Q_device, Q_device_target);
        swap(item_bias_device, item_bias_target);

        cfg->cur_iterations += 1;
    }
    CHECK_CUDA(hipDeviceSynchronize());
    end = clock();

    // Output time taken
    time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;   
    printf("Time taken for %d of iterations is %lf\n", cfg->total_iterations, time_taken);

    // Copy updated P and Q back
    P_device->to_host(P);
    Q_device->to_host(Q);

    // Copy updated bias arrays back
    CHECK_CUDA(hipMemcpy(user_bias, user_bias_device, user_count * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(item_bias, item_bias_device, item_count * sizeof(float), hipMemcpyDeviceToHost));

    // Free memory
    CHECK_CUDA(hipFree(errors_device));
    CHECK_CUDA(hipFree(errors_test_device));
    CHECK_CUDA(hipFree(block_errors_device));
    CHECK_CUDA(hipFree(user_bias_device));
    CHECK_CUDA(hipFree(item_bias_device));
    CHECK_CUDA(hipFree(user_bias_target));
    CHECK_CUDA(hipFree(item_bias_target));
    CHECK_CUDA(hipFree(d_state));
    delete P_device;
    delete P_device_target;
    delete Q_device;
    delete Q_device_target;
    delete [] errors;
    delete [] errors_test;
    delete [] block_errors_host;
}

void train(CudaCSRMatrix* train_matrix, CudaCSRMatrix* test_matrix, config::Config* cfg, float **P_ptr, float **Q_ptr, float **losses_ptr,
           float **user_bias_ptr, float **item_bias_ptr, float global_bias) {
    int item_count = train_matrix->cols;
    // Initialize for regular training
    float *Q = initialize_normal_array(item_count * cfg->n_factors, cfg->n_factors);
    float *item_bias = initialize_normal_array(item_count, cfg->n_factors);
    *Q_ptr = Q;
    *item_bias_ptr = item_bias;
    train(train_matrix, test_matrix, cfg, P_ptr, Q_ptr, Q, losses_ptr, user_bias_ptr, item_bias_ptr, item_bias, global_bias);
}