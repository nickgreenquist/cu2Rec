#include <random>

#include "config.h"
#include "loss.h"
#include "matrix.h"
#include "sgd.h"
#include "util.h"

#define index(i, j, N)  ((i)*(N)) + (j)

using namespace cu2rec;
using namespace std;

void train(CudaCSRMatrix* matrix, config::Config* cfg, float **P_ptr, float **Q_ptr, float **losses_ptr,
           float **user_bias_ptr, float **item_bias_ptr, float global_bias) {
    int user_count = matrix->rows;
    int item_count = matrix->cols;
    cfg->set_cuda_variables();

    // Initialize P and Q
    float *P = initialize_normal_array(user_count * cfg->n_factors);
    float *Q = initialize_normal_array(item_count * cfg->n_factors);
    float *losses = new float[cfg->total_iterations];
    *P_ptr = P;
    *Q_ptr = Q;
    *losses_ptr = losses;

    // Copy P and Q to device memory
    CudaDenseMatrix* P_device = new CudaDenseMatrix(user_count, cfg->n_factors, P);
    CudaDenseMatrix* Q_device = new CudaDenseMatrix(item_count, cfg->n_factors, Q);
    CudaDenseMatrix* P_device_target = new CudaDenseMatrix(user_count, cfg->n_factors, P);
    CudaDenseMatrix* Q_device_target = new CudaDenseMatrix(item_count, cfg->n_factors, Q);

    // Create the errors
    float *errors_device;
    hipMalloc(&errors_device, matrix->nonzeros * sizeof(float));

    // Create the total losses
    float *losses_device;
    hipMalloc(&losses_device, cfg->total_iterations * sizeof(float));
    hipMemset(losses_device, 0, cfg->total_iterations * sizeof(float));

    // Create the bias arrays
    float *user_bias = initialize_normal_array(user_count);
    float *item_bias = initialize_normal_array(item_count);
    *user_bias_ptr = user_bias;
    *item_bias_ptr = item_bias;
    
    float *user_bias_device;
    hipMalloc(&user_bias_device, user_count * sizeof(float));
    hipMemcpy(user_bias_device, user_bias, user_count * sizeof(float), hipMemcpyHostToDevice);

    float *item_bias_device;
    hipMalloc(&item_bias_device, item_count * sizeof(float));
    hipMemcpy(item_bias_device, item_bias, item_count * sizeof(float), hipMemcpyHostToDevice);

    // Create bias targets
    float *user_bias_target, *item_bias_target;
    hipMalloc(&user_bias_target, user_count * sizeof(float));
    hipMemcpy(user_bias_target, user_bias, user_count * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&item_bias_target, item_count * sizeof(float));
    hipMemcpy(item_bias_target, item_bias, item_count * sizeof(float), hipMemcpyHostToDevice);

    // Dimensions
    int n_threads = 32;
    dim3 dim_block(n_threads);
    dim3 dim_grid_sgd(user_count / n_threads + 1);
    dim3 dim_grid_loss(matrix->nonzeros / n_threads + 1);
    dim3 dim_grid_P_reg_loss(P_device->rows * P_device->cols / n_threads + 1);
    dim3 dim_grid_Q_reg_loss(Q_device->rows * Q_device->cols / n_threads + 1);
    dim3 dim_grid_user_bias_reg_loss(user_count / n_threads + 1);
    dim3 dim_grid_item_bias_reg_loss(item_count / n_threads + 1);

    // Training loop
    hipError_t lastError;
    for (int i = 0; i < cfg->total_iterations; ++i) {
        // Calculate initial error per each rating
        calculate_loss_gpu(P_device, Q_device, cfg->n_factors, user_count, item_count, matrix->nonzeros, matrix,
                           errors_device, user_bias_device, item_bias_device, global_bias);

        // Run single iteration of SGD
        sgd_update<<<dim_grid_sgd, dim_block>>>(matrix->indptr, matrix->indices, P_device->data, Q_device->data,
                                                P_device_target->data, Q_device_target->data, errors_device,
                                                user_count, item_count, user_bias_device, item_bias_device,
                                                user_bias_target, item_bias_target);
        lastError = hipGetLastError();
        if(hipSuccess != lastError) {
            printf("ERROR: %s\n", hipGetErrorName(lastError));
        }

        // Calculate total loss to check for improving loss
        total_loss_kernel<<<dim_grid_loss, dim_block>>>(errors_device, losses_device, matrix->nonzeros, i, 1);
        if(cfg->P_reg > 0)
            total_loss_kernel<<<dim_grid_P_reg_loss, dim_block>>>(P_device->data, losses_device, P_device->rows * P_device->cols, i, cfg->P_reg);
        if(cfg->Q_reg > 0)
            total_loss_kernel<<<dim_grid_Q_reg_loss, dim_block>>>(Q_device->data, losses_device, Q_device->rows * Q_device->cols, i, cfg->Q_reg);
        if(cfg->user_bias_reg > 0)
            total_loss_kernel<<<dim_grid_user_bias_reg_loss, dim_block>>>(user_bias_device, losses_device, user_count, i, cfg->user_bias_reg);
        if(cfg->item_bias_reg > 0)
            total_loss_kernel<<<dim_grid_item_bias_reg_loss, dim_block>>>(item_bias_device, losses_device, item_count, i, cfg->item_bias_reg);

        lastError = hipGetLastError();
        if(hipSuccess != lastError) {
            printf("ERROR: %s\n", hipGetErrorName(lastError));
        }

        // The loss kernels modify P, Q, user_bias, and item_bias
        // Copy them back
        // TODO: avoid this entirely
        if(cfg->P_reg > 0)
            hipMemcpy(P_device->data, P_device_target->data, user_count * cfg->n_factors * sizeof(float), hipMemcpyDeviceToDevice);
        if(cfg->Q_reg > 0)
            hipMemcpy(Q_device->data, Q_device_target->data, item_count * cfg->n_factors * sizeof(float), hipMemcpyDeviceToDevice);
        if(cfg->user_bias_reg > 0)
            hipMemcpy(user_bias_device, user_bias_target, user_count * sizeof(float), hipMemcpyDeviceToDevice);
        if(cfg->item_bias_reg > 0)
            hipMemcpy(item_bias_device, item_bias_target, item_count * sizeof(float), hipMemcpyDeviceToDevice);

        lastError = hipGetLastError();
        if(hipSuccess != lastError) {
            printf("ERROR: %s\n", hipGetErrorName(lastError));
        }

        // Swap old and new P and Q
        swap(P_device, P_device_target);
        swap(Q_device, Q_device_target);

        // Swap old and new bias arrays
        swap(user_bias_device, user_bias_target);
        swap(item_bias_device, item_bias_target);

        cfg->cur_iterations += 1;

        // Output current loss for this iteration
        // WARNING - SLOW: Remove after debugging due to hipMemcpy just for printing loss
        if((i + 1) % 10 == 0) {
            hipMemcpy(losses, losses_device, cfg->total_iterations * sizeof(float), hipMemcpyDeviceToHost);
            cout << "Loss for Iteration " << i + 1 << ": " << losses[i] << "\n";
        }
    }
    
    // Copy array of losses back to host
    hipMemcpy(losses, losses_device, cfg->total_iterations * sizeof(float), hipMemcpyDeviceToHost);

    // Copy updated P and Q back
    P_device->to_host(P);
    Q_device->to_host(Q);

    // Copy updated bias arrays back
    hipMemcpy(user_bias, user_bias_device, user_count * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(item_bias, item_bias_device, item_count * sizeof(float), hipMemcpyDeviceToHost);

    // TODO: Remove after debugging issue with outputting components
    // Output final loss and final components to verify results
    float *predictions = new float[user_count * item_count];
    for(int u = 0; u < user_count; u++) {
        const float * p = &P[u * cfg->n_factors];
        for(int i = 0; i < item_count; i++) {
            const float * Qi = &Q[i * cfg->n_factors];
            float pred = global_bias + user_bias[u] + item_bias[i];
            for (int f = 0; f < cfg->n_factors; f++)
                pred += Qi[f]*p[f];
            predictions[index(u, i, item_count)] = pred;
        }
    }
    cout << "Predictions: " <<  "\n";
    for(int u = 0; u < user_count; u++) {
        cout << "[";
        for(int i = 0; i < item_count; i++) {
            cout << predictions[index(u, i, item_count)] << ", ";
        }
        cout << "]\n";
    }

    // Free memory
    hipFree(errors_device);
    hipFree(losses_device);
    hipFree(user_bias_device);
    hipFree(item_bias_device);
    hipFree(user_bias_target);
    hipFree(item_bias_target);
    delete P_device;
    delete P_device_target;
    delete Q_device;
    delete Q_device_target;

    // TODO: remove after debugging issues with outputting components
    delete [] predictions;
}
