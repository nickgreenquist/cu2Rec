#include "hip/hip_runtime.h"
#include <random>
#include <cmath>

#include "config.h"
#include "loss.h"
#include "matrix.h"
#include "sgd.h"
#include "util.h"

#define index(i, j, N)  ((i)*(N)) + (j)

using namespace cu2rec;
using namespace std;

// Inspired by https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
// and https://devblogs.nvidia.com/using-shared-memory-cuda-cc/
// Fixes the problems related to data sizes
float calculate_total_loss(float *in_errors, float *out_errors, float *out_errors_host, int n_errors, int grid_size, int block_size, ErrorType error_type) {
    switch(block_size) {
        case 512:
            total_loss_kernel<512><<<grid_size, block_size, 512 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 256:
            total_loss_kernel<256><<<grid_size, block_size, 256 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 128:
            total_loss_kernel<128><<<grid_size, block_size, 128 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 64:
            total_loss_kernel< 64><<<grid_size, block_size,  64 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 32:
            total_loss_kernel< 32><<<grid_size, block_size,  32 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 16:
            total_loss_kernel< 16><<<grid_size, block_size,  16 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 8:
            total_loss_kernel<  8><<<grid_size, block_size,   8 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 4:
            total_loss_kernel<  4><<<grid_size, block_size,   4 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 2:
            total_loss_kernel<  2><<<grid_size, block_size,   2 * sizeof(float)>>>(in_errors, out_errors, n_errors, error_type);
            break;
        case 1:
            total_loss_kernel<  1><<<grid_size, block_size,   1 * sizeof(float)>>>(in_errors, out_errors, n_errors);
            break;
    }
    hipMemcpy(out_errors_host, out_errors, grid_size * sizeof(float), hipMemcpyDeviceToHost);
    float total = 0;
    for(int k = 0; k < grid_size; k++) {
        total += out_errors_host[k];
    }
    return error_type == RMSE ? sqrt(total / n_errors) : total / n_errors;
}

void train(CudaCSRMatrix* matrix, config::Config* cfg, float **P_ptr, float **Q_ptr, float *Q, float **losses_ptr,
           float **user_bias_ptr, float **item_bias_ptr, float *item_bias, float global_bias) {
    int user_count = matrix->rows;
    int item_count = matrix->cols;
    cfg->set_cuda_variables();

    // Initialize P, Q has already been initialized
    float *P = initialize_normal_array(user_count * cfg->n_factors, cfg->n_factors);
    float *losses = new float[cfg->total_iterations];
    *P_ptr = P;
    *losses_ptr = losses;

    // Copy P and Q to device memory
    CudaDenseMatrix* P_device = new CudaDenseMatrix(user_count, cfg->n_factors, P);
    CudaDenseMatrix* Q_device = new CudaDenseMatrix(item_count, cfg->n_factors, Q);
    CudaDenseMatrix* P_device_target = new CudaDenseMatrix(user_count, cfg->n_factors, P);
    CudaDenseMatrix* Q_device_target = new CudaDenseMatrix(item_count, cfg->n_factors, Q);

    // Create the errors
    float *errors_host = new float[matrix->nonzeros];
    float *errors_device;
    hipMalloc(&errors_device, matrix->nonzeros * sizeof(float));

    // Create the bias array
    float *user_bias = initialize_normal_array(user_count, cfg->n_factors);
    *user_bias_ptr = user_bias;
    
    float *user_bias_device;
    hipMalloc(&user_bias_device, user_count * sizeof(float));
    hipMemcpy(user_bias_device, user_bias, user_count * sizeof(float), hipMemcpyHostToDevice);

    float *item_bias_device;
    hipMalloc(&item_bias_device, item_count * sizeof(float));
    hipMemcpy(item_bias_device, item_bias, item_count * sizeof(float), hipMemcpyHostToDevice);

    // Create bias targets
    float *user_bias_target, *item_bias_target;
    hipMalloc(&user_bias_target, user_count * sizeof(float));
    hipMemcpy(user_bias_target, user_bias, user_count * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&item_bias_target, item_count * sizeof(float));
    hipMemcpy(item_bias_target, item_bias, item_count * sizeof(float), hipMemcpyHostToDevice);

    // Dimensions
    int n_threads = 32;
    dim3 dim_block(n_threads);
    dim3 dim_grid_sgd(user_count / n_threads + 1);
    dim3 dim_grid_loss(256);
    dim3 dim_block_loss(64); // Must be 2^0 to 2^9
    dim3 dim_grid_P_reg_loss(P_device->rows * P_device->cols / n_threads + 1);
    dim3 dim_grid_Q_reg_loss(Q_device->rows * Q_device->cols / n_threads + 1);
    dim3 dim_grid_user_bias_reg_loss(user_count / n_threads + 1);
    dim3 dim_grid_item_bias_reg_loss(item_count / n_threads + 1);

    // Create loss per block
    float *block_errors_host = new float[dim_grid_loss.x];
    float *block_errors_device;
    hipMalloc(&block_errors_device, dim_grid_loss.x * sizeof(float));
    hipMemset(block_errors_device, 0, dim_grid_loss.x * sizeof(float));

    // Create hiprand state
    hiprandState *d_state;
    hipMalloc(&d_state, user_count * sizeof(hiprandState));

    // Training loop
    hipError_t lastError;
    for (int i = 0; i < cfg->total_iterations; ++i) {
        // Calculate initial error per each rating
        calculate_loss_gpu(P_device, Q_device, cfg->n_factors, user_count, item_count, matrix->nonzeros, matrix,
                           errors_device, user_bias_device, item_bias_device, global_bias);

        // Set up random state using iteration as seed
        initCurand<<<dim_grid_sgd, dim_block>>>(d_state, i + 1, user_count);

        // Run single iteration of SGD
        sgd_update<<<dim_grid_sgd, dim_block>>>(matrix->indptr, matrix->indices, P_device->data, Q_device->data,
                                                P_device_target->data, Q_device_target->data, errors_device,
                                                user_count, item_count, user_bias_device, item_bias_device,
                                                user_bias_target, item_bias_target, d_state);
        lastError = hipGetLastError();
        if(hipSuccess != lastError) {
            printf("ERROR: %s\n", hipGetErrorName(lastError));
        }

        // Calculate total loss periodically to check for improving loss
        if((i + 1) % 10 == 0 || i == 0) {
            float rmse = calculate_total_loss(errors_device, block_errors_device, block_errors_host, matrix->nonzeros, dim_grid_loss.x, dim_block_loss.x, RMSE);
            float mae = calculate_total_loss(errors_device, block_errors_device, block_errors_host, matrix->nonzeros, dim_grid_loss.x, dim_block_loss.x, MAE);
            printf("Iteration %d GPU MAE %f RMSE %f\n", i + 1, mae, rmse);
            losses[i] = rmse;
        }
        // if(cfg->P_reg > 0)
        //     total_loss_kernel<<<dim_grid_P_reg_loss, dim_block>>>(P_device->data, losses_device, P_device->rows * P_device->cols, i, cfg->P_reg);
        // if(cfg->Q_reg > 0)
        //     total_loss_kernel<<<dim_grid_Q_reg_loss, dim_block>>>(Q_device->data, losses_device, Q_device->rows * Q_device->cols, i, cfg->Q_reg);
        // if(cfg->user_bias_reg > 0)
        //     total_loss_kernel<<<dim_grid_user_bias_reg_loss, dim_block>>>(user_bias_device, losses_device, user_count, i, cfg->user_bias_reg);
        // if(cfg->item_bias_reg > 0)
        //     total_loss_kernel<<<dim_grid_item_bias_reg_loss, dim_block>>>(item_bias_device, losses_device, item_count, i, cfg->item_bias_reg);

        lastError = hipGetLastError();
        if(hipSuccess != lastError) {
            printf("ERROR: %s\n", hipGetErrorName(lastError));
        }

        // // The loss kernels modify P, Q, user_bias, and item_bias
        // // Copy them back
        // // TODO: avoid this entirely
        // if(cfg->P_reg > 0)
        //     hipMemcpy(P_device->data, P_device_target->data, user_count * cfg->n_factors * sizeof(float), hipMemcpyDeviceToDevice);
        // if(cfg->Q_reg > 0)
        //     hipMemcpy(Q_device->data, Q_device_target->data, item_count * cfg->n_factors * sizeof(float), hipMemcpyDeviceToDevice);
        // if(cfg->user_bias_reg > 0)
        //     hipMemcpy(user_bias_device, user_bias_target, user_count * sizeof(float), hipMemcpyDeviceToDevice);
        // if(cfg->item_bias_reg > 0)
        //     hipMemcpy(item_bias_device, item_bias_target, item_count * sizeof(float), hipMemcpyDeviceToDevice);

        // lastError = hipGetLastError();
        // if(hipSuccess != lastError) {
        //     printf("ERROR: %s\n", hipGetErrorName(lastError));
        // }

        // Swap old and new P and Q
        swap(P_device, P_device_target);
        swap(Q_device, Q_device_target);

        // Swap old and new bias arrays
        swap(user_bias_device, user_bias_target);
        swap(item_bias_device, item_bias_target);

        cfg->cur_iterations += 1;
    }
    

    // Copy updated P and Q back
    P_device->to_host(P);
    Q_device->to_host(Q);

    // Copy updated bias arrays back
    hipMemcpy(user_bias, user_bias_device, user_count * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(item_bias, item_bias_device, item_count * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(errors_device);
    hipFree(block_errors_device);
    hipFree(user_bias_device);
    hipFree(item_bias_device);
    hipFree(user_bias_target);
    hipFree(item_bias_target);
    hipFree(d_state);
    delete P_device;
    delete P_device_target;
    delete Q_device;
    delete Q_device_target;
    delete [] errors_host;
    delete [] block_errors_host;
}

void train(CudaCSRMatrix* matrix, config::Config* cfg, float **P_ptr, float **Q_ptr, float **losses_ptr,
           float **user_bias_ptr, float **item_bias_ptr, float global_bias) {
    int item_count = matrix->cols;
    // Initialize for regular training
    float *Q = initialize_normal_array(item_count * cfg->n_factors, cfg->n_factors);
    float *item_bias = initialize_normal_array(item_count, cfg->n_factors);
    *Q_ptr = Q;
    *item_bias_ptr = item_bias;
    train(matrix, cfg, P_ptr, Q_ptr, Q, losses_ptr, user_bias_ptr, item_bias_ptr, item_bias, global_bias);
}
